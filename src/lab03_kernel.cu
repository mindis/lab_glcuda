#include <hip/hip_runtime.h>


/* initialize grid
 * we'll distribute all particles evenly on the screen
 */
__global__ void d_reset( float4* verts, float4* states,
                         float ww, float wh,
                         int mesh_width, int mesh_height
  )
{
  int x, y;
  for (y = blockIdx.y * blockDim.y + threadIdx.y;
       y < mesh_height;
       y += blockDim.y * gridDim.y)
  {
    for (x = blockIdx.x * blockDim.x + threadIdx.x;
         x < mesh_width;
         x += blockDim.x * gridDim.x)
    {
      int vi = y*mesh_width + x;
      float u, v, w;
      u = ww * (-0.5f + x*1.f/mesh_width);
      v = wh * (-0.5f + y*1.f/mesh_height);
      w = 0.0f;
      verts[vi].x = u;
      verts[vi].y = v;
      verts[vi].z = w;
      states[vi].x = 0.0f;
      states[vi].y = 0.0f;
      states[vi].z = 0.0f;
      /* color information is uploaded by host, no init here */
    }
  }
}

/*
 * calculate particle forces & new positions & colors
 */
__global__ void d_advance( float4* verts, float4* states,
                           float mx, float my,
                           int mesh_count,
                           float speed,
                           float delta)
{
  int vi;
  for (vi = blockIdx.x * blockDim.x + threadIdx.x;
       vi < mesh_count;
       vi += blockDim.x * gridDim.x)
  {
    // colors are stored after all vertices
    unsigned int ci = mesh_count + vi;

    float3 state  = {states[vi].x, states[vi].y, 0.f/*states[vi].z*/};
    // calculate vector between mouse and particle
    float3 dir_force = {verts[vi].x-mx, verts[vi].y-my, 0.f};
    /* steering:
     * the new particle state vector is given by current directional force
     * and the old state scaled by the particle mass
     */
    float flen = rsqrtf( dir_force.x*dir_force.x
                         +dir_force.y*dir_force.y
                         /*+dir_force.z*dir_force.z*/);
    dir_force.x *= flen;
    dir_force.y *= flen;
    /*dir_force.z *= flen;*/
    state.x += delta * dir_force.x;
    state.y += delta * dir_force.y;
    /*state.z += dir_force.z;*/

    // speed: base speed scale by particle mass
    float v = speed / states[vi].w;
    /* update particle position:
     * new_position = old_position - state_vector*speed;
     */
    float3 dv = {state.x * v, state.y * v, 0.f/*state.z * v*/};

    verts[vi].x -= delta * dv.x;
    verts[vi].y -= delta * dv.y;
    /*verts[vi].z -= delta * dv.z;*/
    // update colors depending on particles force
    float dx = 0.5f*dv.x;
    float dy = 0.5f*dv.y;
    verts[ci].x = 0.1f+fabs(dx);
    verts[ci].y = 0.1f+fabs(dy);
    verts[ci].z = 0.1f+fabs(dx+dy);
    states[vi].x = state.x;
    states[vi].y = state.y;
    /*states[vi].z = state.z;*/
  }
}

void kernel_reset(float4* verts, float4* states,
                  int ww, int wh,
                  int mesh_width, int mesh_height,
                  int numSMs)
{
  dim3 threads(16,16);
  dim3 blocks(16*numSMs);

  d_reset<<<blocks, threads>>>( verts, states,
                                ww, wh,
                                mesh_width, mesh_height );
}

void kernel_advance(float4* verts, float4* states,
                    float mx, float my,
                    int mesh_count,
                    float speed,
                    int numSMs,
                    double delta)
{
  dim3 threads(128);
  dim3 blocks( 16*numSMs );
    
  d_advance<<<blocks, threads>>>( verts, states,
                                  mx, my,
                                  mesh_count,
                                  speed,
                                  static_cast<float>(delta));
}
